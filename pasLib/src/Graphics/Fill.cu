#include "hip/hip_runtime.h"
#include "DrawingOperations.h"
#include "CuUtils.hx"
namespace pasLib
{
    int n=0;
    __global__ void K_Fill(int color, int wid, int hei, int* image)
    {
        int i = blockIdx.y*blockDim.y + threadIdx.y;
		int j = blockIdx.x*blockDim.x + threadIdx.x;
        if (i < wid && j < hei)
        {
            //*(image + i*wid+j) = color;
        }
    }
    
    void Fill(int color)
    {
        n++;
        int wid, hei;
        int* buf;
        GetWindowBufferInfo(&wid, &hei, &buf);
        int num_blocks_w = (wid  + (BLOCK_SIZE-1))/BLOCK_SIZE;
        int num_blocks_h = (hei + (BLOCK_SIZE-1))/BLOCK_SIZE;
        dim3 gridConf(num_blocks_w, num_blocks_h);
        dim3 blockConf(BLOCK_SIZE, BLOCK_SIZE);
        if (n==50)
        {
            __sdump(wid);
            __sdump(hei);
            __sdump(color);
            __sdump(buf);
            __sdump(num_blocks_w);
            __sdump(num_blocks_h);
            __sdump(BLOCK_SIZE);
            K_Fill<<<gridConf, blockConf>>>(color, wid, hei, buf);
            CU_CHK(hipGetLastError());
        }
            
    }
}